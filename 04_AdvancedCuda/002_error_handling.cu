#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c)
{
	*d_c = *d_a + *d_b;
}

int main()
{
	//Defining host variables
	int h_a, h_b, h_c;
	//Defining Device Pointers
	int* d_a, * d_b, * d_c;
	//Initializing host variables
	h_a = 1;
	h_b = 4;

	hipError_t cudaStatus;

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&d_c, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}
	cudaStatus = hipMalloc((void**)&d_a, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&d_b, sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_a, &h_a, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_b, &h_b, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	gpuAdd << <1, 1 >> > (d_a, d_b, d_c);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	printf("Passing Parameter by Reference Output: %d + %d = %d\n", h_a, h_b, h_c);
Error:
	hipFree(d_c);
	hipFree(d_a);
	hipFree(d_b);

	return 0;
}