#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#define N 50000

__global__ void gpuAdd(int* d_a, int* d_b, int* d_c)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		tid += blockDim.x + gridDim.x;
	}
}

int main()
{
	int* h_a, * h_b, * h_c;
	int* d_a0, * d_b0, * d_c0;
	int* d_a1, * d_b1, * d_c1;

	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);

	hipEvent_t e_start, e_stop;
	hipEventCreate(&e_start);
	hipEventCreate(&e_stop);
	hipEventRecord(e_start, 0);

	// Allocate pinned memory on the host
	hipHostAlloc((void**)&h_a, N * 2 * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&h_b, N * 2 * sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&h_c, N * 2 * sizeof(int), hipHostMallocDefault);

	hipMalloc((void**)&d_a0, N * sizeof(int));
	hipMalloc((void**)&d_b0, N * sizeof(int));
	hipMalloc((void**)&d_c0, N * sizeof(int));
	hipMalloc((void**)&d_a1, N * sizeof(int));
	hipMalloc((void**)&d_b1, N * sizeof(int));
	hipMalloc((void**)&d_c1, N * sizeof(int));

	for (int i = 0; i < N * 2; i++)
	{
		h_a[i] = 2 * i * i;
		h_b[i] = i;
	}

	hipMemcpyAsync(d_a0, h_a, N * sizeof(int), hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(d_a1, h_a + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
	hipMemcpyAsync(d_b0, h_b, N * sizeof(int), hipMemcpyHostToDevice, stream0);
	hipMemcpyAsync(d_b1, h_b + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);

	gpuAdd << <512, 512, 0, stream0 >> > (d_a0, d_b0, d_c0);
	gpuAdd << <512, 512, 0, stream1 >> > (d_a1, d_b1, d_c1);

	hipMemcpyAsync(h_c, d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
	hipMemcpyAsync(h_c + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream0);

	hipDeviceSynchronize();
	hipEventRecord(e_stop, 0);
	hipEventSynchronize(e_stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, e_start, e_stop);
	printf("Time to add %d numbers: %3.1f ms\n", 2 * N, elapsedTime);

	int Correct = 1;
	printf("Vector addition on GPU \n");
	for (int i = 0; i < 2 * N; i++) {
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
			Correct = 0;
		}
	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
		printf("There is an Error in GPU Computation\n");
	}

	hipFree(d_a0);
	hipFree(d_b0);
	hipFree(d_c0);
	hipFree(d_a1);
	hipFree(d_b1);
	hipFree(d_c1);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	hipStreamDestroy(stream0);
	hipStreamDestroy(stream1);
	return 0;
}