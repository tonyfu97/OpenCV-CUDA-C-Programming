#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>

#define ARRAY_SIZE 5
#define THREAD_PER_BLOCK 5

__global__ void rankSort(int* d_a, int* d_b)
{
	int count = 0;
	int tid = threadIdx.x;
	int ttid = blockIdx.x * THREAD_PER_BLOCK + tid;
	int val = d_a[ttid];

	__shared__ int cache[THREAD_PER_BLOCK];
	for (int i = tid; i < ARRAY_SIZE; i+=THREAD_PER_BLOCK)
	{
		cache[tid] = d_a[i];
		__syncthreads();
		for (int j = 0; j < THREAD_PER_BLOCK; j++)
		{
			if (val > cache[j])
			{
				count++;
			}
			__syncthreads();
		}
	}
	d_b[count] = val;
}

int main()
{
	int h_a[ARRAY_SIZE] = { 5, 9, 3, 4, 8 };
	int h_b[ARRAY_SIZE];
	int* d_a, * d_b;

	hipMalloc((void**)&d_a, ARRAY_SIZE * sizeof(int));
	hipMalloc((void**)&d_b, ARRAY_SIZE * sizeof(int));
	
	hipMemcpy(d_a, h_a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);

	rankSort << <(ARRAY_SIZE + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>> (d_a, d_b);

	// hipDeviceSynchronize(); This is not neccessary

	hipMemcpy(h_b, d_b, ARRAY_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	printf("The sorted array is : \n");
	for (int i = 0; i < ARRAY_SIZE - 1; i++)
	{
		printf("%d, ", h_b[i]);
	}
	printf("%d\n", h_b[ARRAY_SIZE - 1]);

	hipFree(d_a);
	hipFree(d_b);
	return 0;
}