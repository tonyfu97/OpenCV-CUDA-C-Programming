#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define TILE_SIZE 2

__global__ void gpu_matmul_nonshared(float* d_a, float* d_b, float* d_c, const int size)
{
	int col = TILE_SIZE * blockIdx.x + threadIdx.x;
	int row = TILE_SIZE * blockIdx.y + threadIdx.y;

	for (int k = 0; k < size; k++)
	{
		d_c[row * size + col] += d_a[row * size + k] * d_b[k * size + col];
	}
}

__global__ void gpu_matmul_shared(float* d_a, float* d_b, float* d_c, const int size)
{
	__shared__ float shared_a[TILE_SIZE][TILE_SIZE];
	__shared__ float shared_b[TILE_SIZE][TILE_SIZE];

	int col = TILE_SIZE * blockIdx.x + threadIdx.x;
	int row = TILE_SIZE * blockIdx.y + threadIdx.y;

	for (int i = 0; i < size / TILE_SIZE; i++)
	{
		shared_a[threadIdx.y][threadIdx.x] = d_a[row * size + (i * TILE_SIZE + threadIdx.x)];
		shared_b[threadIdx.y][threadIdx.x] = d_b[(i * TILE_SIZE + threadIdx.y) * size + col];

		__syncthreads();

		for (int j = 0; j < TILE_SIZE; j++)
		{
			d_c[row * size + col] += shared_a[threadIdx.y][j] * shared_b[j][threadIdx.x];
		}
		__syncthreads();
	}
}

int main()
{
	const int size = 4;
	float h_a[size][size], h_b[size][size], h_result[size][size];
	float *d_a, *d_b, *d_result;

	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			h_a[i][j] = i;
			h_b[i][j] = j;
		}
	}
	
	hipMalloc((void**)&d_a, size * size * sizeof(int));
	hipMalloc((void**)&d_b, size * size * sizeof(int));
	hipMalloc((void**)&d_result, size * size * sizeof(int));

	hipMemcpy(d_a, h_a, size * size * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size * size * sizeof(int), hipMemcpyHostToDevice);

	dim3 dimGrid(size / TILE_SIZE, size / TILE_SIZE, 1);
	dim3 dimBlock(TILE_SIZE, TILE_SIZE, 1);

	//gpu_matmul_nonshared << <dimGrid, dimBlock >> > (d_a, d_b, d_result, size);

	gpu_matmul_shared << <dimGrid, dimBlock >> > (d_a, d_b, d_result, size);

	hipMemcpy(h_result, d_result, size * size * sizeof(int), hipMemcpyDeviceToHost);
	printf("The result of Matrix multiplication is: \n");

	for (int i = 0; i < size; i++)
	{
		for (int j = 0; j < size; j++)
		{
			printf("%f   ", h_result[i][j]);
		}
		printf("\n");
	}
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_result);
	return 0;
}