#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define N 5

__constant__ int constant_f;
__constant__ int constant_g;

__global__ void gpu_constant_memory(float* d_in, float* d_out)
{
	int tid = threadIdx.x;
	d_out[tid] = constant_f * d_in[tid] + constant_g;
}

int main()
{
	float h_in[N], h_out[N];
	float* d_in, * d_out;

	int h_f = 2;
	int h_g = 20;

	hipMalloc((void**)&d_in, N * sizeof(float));
	hipMalloc((void**)&d_out, N * sizeof(float));

	for (int i = 0; i < N; i++)
	{
		h_in[i] = i;
	}

	hipMemcpy(d_in, h_in, N * sizeof(float), hipMemcpyHostToDevice);

	// Important! Copy constants to memory
	hipMemcpyToSymbol(HIP_SYMBOL(constant_f), &h_f, sizeof(int), 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(constant_g), &h_g, sizeof(int), 0, hipMemcpyHostToDevice);

	// Kernel call
	gpu_constant_memory << <1, N >> > (d_in, d_out);

	hipMemcpy(h_out, d_out, N * sizeof(float), hipMemcpyDeviceToHost);

	printf("Use of Constant memory on GPU \n");
	for (int i = 0; i < N; i++) {
		printf("The expression for input %f is %f\n", h_in[i], h_out[i]);
	}
	//Free up memory
	hipFree(d_in);
	hipFree(d_out);
	return 0;
}