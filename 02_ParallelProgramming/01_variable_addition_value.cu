#include <hip/hip_runtime.h>

#include <iostream>


__global__ void gpuAdd(int d_a, int d_b, int* d_c)
{
	*d_c = d_a + d_b;
}

int main()
{
	int h_c; // Host variable to store answer
	int* d_c; // Device pointer

	hipMalloc((void**)&d_c, sizeof(int));
	gpuAdd <<<1, 1 >>> (1, 4, d_c);

	hipMemcpy(&h_c, d_c, sizeof(int), hipMemcpyDeviceToHost);
	printf("1 + 4 = %d/n", h_c);

	hipFree(d_c);
	return 0;
}